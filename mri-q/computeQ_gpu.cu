#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
#include "support.h"

#define PI   3.1415926535897932384626433832795029f
#define PIx2 6.2831853071795864769252867665590058f

#define MIN(X,Y) ((X) < (Y) ? (X) : (Y))
#define K_ELEMS_PER_GRID 2048

#define PHIMAGBLOCK_SIZE 512 // 512 or 192

struct kValues {
  float Kx;
  float Ky;
  float Kz;
  float PhiMag;
};



__global__ void ComputePhiMagGPUKernel(int numk, float* phiR, float* phiI, float* phiMag){

  unsigned int t = threadIdx.x;
  unsigned int offset = (blockIdx.x*blockDim.x) + t;

  // __shared__ float real[blockDim.x];
  // __shared__ float imag[blockDim.x];
  // __shared__ float phiOut[blockDim.x];

  if(offset < numk){
    float real = phiR[offset];
    float imag = phiI[offset];
    phiMag[offset] = real*real + imag*imag;
  }


}


// (TODO) This is actually slower????
inline
void 
ComputePhiMagGPU(int numK, float* phiR, float* phiI, float* phiMag) {
  hipError_t cuda_ret;

  float *phiR_d, *phiI_d, *phiMag_d;

  // Allocate device variables ---------------------------------
  cuda_ret = hipMalloc((void**)&phiR_d, numK * sizeof(float));
  if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory ");
  cuda_ret = hipMalloc((void**)&phiI_d, numK * sizeof(float));
  if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory ");
  cuda_ret = hipMalloc((void**)&phiMag_d, numK * sizeof(float));
  if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory ");

  cuda_ret = hipMemcpy(phiR_d, phiR, numK * sizeof(float), hipMemcpyHostToDevice);
  if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device ");
  cuda_ret = hipMemcpy(phiI_d, phiI, numK * sizeof(float), hipMemcpyHostToDevice);
  if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device ");
  cuda_ret = hipMemset(phiMag_d, 0, numK * sizeof(float));
  if(cuda_ret != hipSuccess) FATAL("Unable to set device memory ");


  // Launch kernel ----------------------------------------------------------

  dim3 dim_grid, dim_block;
  unsigned block, grid;
  block = PHIMAGBLOCK_SIZE;
  grid = numK / (PHIMAGBLOCK_SIZE);
  if( numK % (PHIMAGBLOCK_SIZE * grid)) grid++;

  printf("\tBLOCK: %d\n\tGRID: %d\n", block, grid);

  dim_block.x = block;
  dim_block.y = 1;
  dim_block.z = 1;

  dim_grid.x = grid;
  dim_grid.y = 1;
  dim_grid.z = 1;

  ComputePhiMagGPUKernel<<<dim_grid, dim_block>>>(numK, phiR_d, phiI_d, phiMag_d);

  cuda_ret = hipDeviceSynchronize();
  if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");


  cuda_ret = hipMemcpy(phiMag, phiMag_d, numK * sizeof(float), hipMemcpyDeviceToHost);
  if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host in naive reduction.");

  hipFree(phiR_d);
  hipFree(phiI_d);
  hipFree(phiMag_d);
}

inline
void 
ComputePhiMagCPU(int numK, float* phiR, float* phiI, float* phiMag) {
  // Baseline Implementation
  int indexK = 0;
  for (indexK = 0; indexK < numK; indexK++) {
    float real = phiR[indexK];
    float imag = phiI[indexK];
    phiMag[indexK] = real*real + imag*imag;
  }
}



// __global__ void ComputeQGPUKernel_Risky(int numK, int numX, struct kValues *kVals, float* x, float* y, float* z, float *__restrict__ Qr, float *__restrict__ Qi){
// }
// void ComputeQGPU_Risky(int numK, int numX, struct kValues *kVals, float* x, float* y, float* z, float *__restrict__ Qr, float *__restrict__ Qi){
// }

__global__ void ComputeQGPUKernel(int numK, int numX, struct kValues *kVals, float* x, float* y, float* z, float *__restrict__ Qr, float *__restrict__ Qi){
  __shared__ float x_s[PHIMAGBLOCK_SIZE];
  __shared__ float y_s[PHIMAGBLOCK_SIZE];
  __shared__ float z_s[PHIMAGBLOCK_SIZE];
  __shared__ struct kValues kVals_s[PHIMAGBLOCK_SIZE];

  unsigned int t = threadIdx.x;
  unsigned int offset = (blockIdx.x*PHIMAGBLOCK_SIZE) + t;


  if(offset < numX){
    x_s[t] = x[offset];
    y_s[t] = y[offset];
    z_s[t] = z[offset];
    kVals_s[t] = kVals[offset];

    int indexK;
    float Qracc = 0.0f;
    float Qiacc = 0.0f;
    float expArg;
    float cosArg;
    float sinArg;

    for (indexK = 0; indexK < numK; indexK++) {
      expArg = PIx2 * (kVals_s[indexK].Kx * x_s[offset] + kVals_s[indexK].Ky * y_s[offset] + kVals_s[indexK].Kz * z_s[offset]);

      cosArg = cosf(expArg);
      sinArg = sinf(expArg);

      float phi = kVals[indexK].PhiMag;
      Qracc += phi * cosArg;
      Qiacc += phi * sinArg;
    }
    Qr[offset] = Qracc;
    Qi[offset] = Qiacc;

  }

}

void ComputeQGPU(int numK, int numX, struct kValues *kVals, float* x, float* y, float* z, float *__restrict__ Qr, float *__restrict__ Qi){
  hipError_t cuda_ret;

  float *x_d, *y_d, *z_d, *Qr_d, *Qi_d;
  struct kValues *kVals_d;

  // Allocate device variables ---------------------------------
  cuda_ret = hipMalloc((void**)&x_d, numX * sizeof(float));
  if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory ");
  cuda_ret = hipMalloc((void**)&y_d, numX * sizeof(float));
  if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory ");
  cuda_ret = hipMalloc((void**)&z_d, numX * sizeof(float));
  if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory ");
  cuda_ret = hipMalloc((void**)&Qr_d, numK * sizeof(float));
  if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory ");
  cuda_ret = hipMalloc((void**)&Qi_d, numK * sizeof(float));
  if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory ");
  cuda_ret = hipMalloc((void**)&kVals_d, numK * sizeof(struct kValues));
  if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory ");


  cuda_ret = hipMemcpy(x_d, x, numX * sizeof(float), hipMemcpyHostToDevice);
  if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device ");
  cuda_ret = hipMemcpy(y_d, y, numX * sizeof(float), hipMemcpyHostToDevice);
  if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device ");
  cuda_ret = hipMemcpy(z_d, z, numX * sizeof(float), hipMemcpyHostToDevice);
  if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device ");
  cuda_ret = hipMemset(Qr_d, 0, numK * sizeof(float));
  if(cuda_ret != hipSuccess) FATAL("Unable to set device memory ");
  cuda_ret = hipMemset(Qi_d, 0, numK * sizeof(float));
  if(cuda_ret != hipSuccess) FATAL("Unable to set device memory ");
  cuda_ret = hipMemcpy(kVals_d, kVals, numK * sizeof(struct kValues), hipMemcpyHostToDevice);
  if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device ");


  // Launch kernel ----------------------------------------------------------

  dim3 dim_grid, dim_block;
  unsigned block, grid;
  block = PHIMAGBLOCK_SIZE;
  grid = numX / (PHIMAGBLOCK_SIZE);
  if( numX % (PHIMAGBLOCK_SIZE * grid)) 
    grid++;

  printf("\tBLOCK: %d\n\tGRID: %d\n", block, grid);

  dim_block.x = block;
  dim_block.y = 1;
  dim_block.z = 1;

  dim_grid.x = grid;
  dim_grid.y = 1;
  dim_grid.z = 1;

  ComputeQGPUKernel<<<dim_grid, dim_block>>>(numK, numX, kVals_d, x_d, y_d, z_d, Qr_d, Qi_d);




  cuda_ret = hipMemcpy(Qr, Qr_d, numK * sizeof(float), hipMemcpyDeviceToHost);
  if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host in naive reduction.");
  cuda_ret = hipMemcpy(Qi, Qi_d, numK * sizeof(float), hipMemcpyDeviceToHost);
  if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host in naive reduction.");

  hipFree(x_d);
  hipFree(y_d);
  hipFree(z_d);
  hipFree(Qr_d);
  hipFree(Qi_d);
  hipFree(kVals_d);

  cuda_ret = hipDeviceSynchronize();
  if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");

}


inline
void
ComputeQCPU(int numK, int numX, struct kValues *kVals, float* x, float* y, float* z, float *__restrict__ Qr, float *__restrict__ Qi) {
  float expArg;
  float cosArg;
  float sinArg;

  int indexK, indexX;

  // Loop over the space and frequency domains.
  // Generally, numX > numK.
  // Since loops are not tiled, it's better that the loop with the smaller
  // cache footprint be innermost.
  for (indexX = 0; indexX < numX; indexX++) {

    // Sum the contributions to this point over all frequencies
    float Qracc = 0.0f;
    float Qiacc = 0.0f;
    for (indexK = 0; indexK < numK; indexK++) {
      expArg = PIx2 * (kVals[indexK].Kx * x[indexX] +
                       kVals[indexK].Ky * y[indexX] +
                       kVals[indexK].Kz * z[indexX]);

      cosArg = cosf(expArg);
      sinArg = sinf(expArg);

      float phi = kVals[indexK].PhiMag;
      Qracc += phi * cosArg;
      Qiacc += phi * sinArg;
    }
    Qr[indexX] = Qracc;
    Qi[indexX] = Qiacc;
  }
}

void createDataStructsCPU(int numK, int numX, float** phiMag, float** Qr, float** Qi){
  *phiMag = (float* ) memalign(16, numK * sizeof(float));
  *Qr = (float*) memalign(16, numX * sizeof (float));
  memset((void *)*Qr, 0, numX * sizeof(float));
  *Qi = (float*) memalign(16, numX * sizeof (float));
  memset((void *)*Qi, 0, numX * sizeof(float));
}





















